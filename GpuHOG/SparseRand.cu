#include "hip/hip_runtime.h"
#include "mex.h"
#include "hip/hip_runtime.h"
#include "hipsparse.h"
#include "hiprand.h"
#include "stdlib.h"
#include <algorithm>
#include "hiprand/hiprand_kernel.h"
#include "time.h"
#define HOG_THREAD_CNT 1
#define NITER 200

using namespace std;

struct Params {
float *Aval;
int *ArowCSR;
int *Acol;
float *b;
float *x;
int N;
int D;
float *devData;
hiprandState *state;
};
//KERNEL CODE FOR RANDOM NUMBER GENERATION
__global__ void  generate_normal_kernel(struct Params params)
{
int i,j,k;
int e;
float grad;
hiprandState localState;


for (i = threadIdx.x; i < HOG_THREAD_CNT; i += HOG_THREAD_CNT)
 {
        localState = params.state[threadIdx.x];
        for (j = 0; j < NITER ; j ++) 
        { 
            e = floorf(hiprand_uniform(&localState)*params.N);
            //params.devData[i*NITER + j]=e; 
            grad = 0.0;
            for(k = params.ArowCSR[e]; k<params.ArowCSR[e+1]; k++)
            {
               grad += ((params.Aval[k])*(params.x[params.Acol[k]]));
            }
            grad = grad - params.b[e];                                // params.Aval[params.ArowCSR[e-1]-1] corresponds to the first element of e-th row
            //params.devData[i*NITER + j] = grad;
            
            for(k = params.ArowCSR[e]; k<params.ArowCSR[e+1]; k++)
            {
                params.x[params.Acol[k]] = params.x[params.Acol[k]] - 0.1*(grad*params.Aval[k] +   0.1*params.x[params.Acol[k]]);
            }    
           
            //params.devData[i*NITER + j] = params.x[params.Acol[k]] - 1.0*(grad*params.Aval[k] +   1.0*params.x[params.Acol[k]]);
        }     
 }
} 

// KERNEL CODE FOR RANDOM NUMBER SEED INITIALIZATION
__global__ void setup_kernel(hiprandState *state, unsigned int seed)
{
    int id = threadIdx.x;
    /* Each thread gets different seed, a different sequence
       number, no offset */
    hiprand_init(seed, id, 0, &state[id]);
}

void mexFunction( int nlhs, mxArray *plhs[],
 int nrhs, const mxArray *prhs[])
{
 struct Params funcparams;
 int dims0[1];//[2];    // For storing matrix dimensions
 int nnz,D,N;         // nnz: Number of non-zero elements in A; D: dimensions of A
 // Allocate memory for storing the matrix x,A and b
 float *x = 0;          // CPU
 float* b = 0;
 float* Aval = 0;
 int* Acol = 0;
 int* Arow = 0;
 int* ArowCSR = 0;
 
 float* gpub = 0;     // GPU
 float* gpux = 0;
 float* gpuAval = 0;       
 int* gpuAcol = 0;
 int* gpuArow = 0;
 int* gpuArowCSR = 0;
 // Allocate memory of random number generation
 hiprandState *state;
 float *hostData,*devData;
 // Initialize Cusparse
 hipsparseStatus_t status;
 hipsparseHandle_t handle=0;
 hipsparseMatDescr_t descr=0;
 
 // Validate Inputs
 if (nrhs != 4) {
 mexErrMsgTxt("engine requires 4 input arguments");
 } else if (nlhs != 1) {
 mexErrMsgTxt("engine requires 1 output argument");
 }
 if ( !mxIsSingle(prhs[2])|| !mxIsSingle(prhs[3]) ) {
 mexErrMsgTxt("A and b must be single precision");
 }
 if ( !mxIsUint32(prhs[0])|| !mxIsUint32(prhs[0]) ) {
 mexErrMsgTxt("Rows and colums of the sparse matrix should be unsigned integers");
 }
 // Get the various dimensions to this problem.
 nnz = mxGetM(prhs[0]); /* Number of nnz in sparse matrix */
 // Fetch Inputs
 Arow =  (int*) mxGetData(prhs[0]);
 Acol =  (int*) mxGetData(prhs[1]);
 Aval =  (float*) mxGetData(prhs[2]);
 D = *max_element(Acol, Acol+nnz) + 1;
 N = *max_element(Arow, Arow+nnz) + 1;
 mexPrintf("N = %d, D = %d \n",N,D);
 b = (float*) mxGetData(prhs[3]);
 
 //This section is for testing
 for (int i=0; i<nnz; i++)
    mexPrintf("%d \t %d \t %f \n ", Arow[i], Acol[i], Aval[i]);
 
 dims0[0]=D;
 //dims0[1]=2*NX;
 plhs[0] = mxCreateNumericArray(1,dims0,mxSINGLE_CLASS,mxREAL);
 x = (float*) mxGetData(plhs[0]);
 // Set random seed for initializing x
 srand (time(NULL));
 for(int i = 0; i<D ; i++)
    x[i] = (float)(i+1);//x[i] = (rand()%100)/100.0; //testing phase
// Testing section ends here
 
 // Allocating space on the cpu for random numbers
 hostData = (float*) malloc ((HOG_THREAD_CNT)*NITER*sizeof(hostData[0]));
  
 // Allocating space on GPU for random Numbers
 hipMalloc((void **) &devData,HOG_THREAD_CNT*NITER*sizeof(devData[0]));
 hipMalloc((void **)&state, HOG_THREAD_CNT*sizeof(state[0]));
  
 // Allocating Aval, Acol, Arow, ArowCSR on GPU
 hipMalloc ((void **)&gpuAval, nnz*sizeof(gpuAval[0]));
 hipMalloc ((void **)&gpuAcol, nnz*sizeof(gpuAcol[0]));
 hipMalloc ((void **)&gpuArow, nnz*sizeof(gpuArow[0]));
 hipMalloc ((void **)&gpuArowCSR, (N+1)*sizeof(gpuArowCSR[0]));
 hipMalloc ((void **)&gpub, N*sizeof(gpub[0]));
 hipMalloc ((void **)&gpux, D*sizeof(gpux[0]));
 // Copying Aval, Acol, Arow from host to device
 hipMemcpy (gpuAval, Aval, nnz*sizeof(Aval[0]), hipMemcpyHostToDevice);
 hipMemcpy (gpuAcol, Acol, nnz*sizeof(Acol[0]), hipMemcpyHostToDevice);
 hipMemcpy (gpuArow, Arow, nnz*sizeof(Arow[0]), hipMemcpyHostToDevice);
 hipMemcpy (gpub, b, N*sizeof(b[0]), hipMemcpyHostToDevice);
 hipMemcpy (gpux, x, D*sizeof(x[0]), hipMemcpyHostToDevice);
 
 /* initialize cusparse library */ 
 status= hipsparseCreate(&handle); 
 if (status != HIPSPARSE_STATUS_SUCCESS)
 mexPrintf("CUSPARSE Library initialization failed"); 
 /* create and setup matrix descriptor */ 
 status= hipsparseCreateMatDescr(&descr);
 if (status != HIPSPARSE_STATUS_SUCCESS)
 mexPrintf("Matrix descriptor initialization failed"); 
 hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
 hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO); 
 // Convert from COO to CSR 
 status = hipsparseXcoo2csr(handle,gpuArow,nnz,N,gpuArowCSR,HIPSPARSE_INDEX_BASE_ZERO);
 if (status != HIPSPARSE_STATUS_SUCCESS) 
    mexPrintf("Conversion from COO to CSR format failed");
 
 //Copy the results of conversion
 mexPrintf("No errors so far");
 ArowCSR = (int*) malloc ((N+1)*sizeof(ArowCSR[0]));
 hipMemcpy (ArowCSR, gpuArowCSR, (size_t)(N+1)*sizeof(ArowCSR[0]), hipMemcpyDeviceToHost);
// if (status != hipSuccess) 
 //   mexPrintf("Copying COO to CSR to HOST  failed");

 mexPrintf("Printing CSR Values \n");
 for (int i =0; i<(N+1); i++)
     mexPrintf("Index=%d, Value = %d \t",i,ArowCSR[i]);
//    mexPrintf("%d \t",ArowCSR[i]);
 
 // Set up Kernel for random number seed initializaton
 setup_kernel<<<1,HOG_THREAD_CNT>>>(state,time(NULL));
 mexPrintf("RANDOM NUMBER SEED INITIALIZED \n");
 
 funcparams.Aval = gpuAval;
 funcparams.ArowCSR = gpuArowCSR;
 funcparams.Acol = gpuAcol;
 funcparams.b = gpub;
 funcparams.x = gpux;
 funcparams.N = N;
 funcparams.D = D;
 funcparams.devData = devData;
 funcparams.state = state;

 
 // Generate Random Numbers
 generate_normal_kernel<<<1, HOG_THREAD_CNT>>>(funcparams);
 hipMemcpy (hostData, devData, HOG_THREAD_CNT*NITER*sizeof(hostData[0]), hipMemcpyDeviceToHost);
 
 // displaying results
 
 for(int i =0; i < HOG_THREAD_CNT*NITER ; i++) {
        if(i%NITER == 0)
            mexPrintf("\n");
        mexPrintf("%f\t",hostData[i]);
    }


//hipMalloc ((void **)&cosRes, 2*NN*sizeof(cosRes[0]));
/* hipMalloc ((void **)&Aarg, N*D*sizeof(Aarg[0]));
 hipMalloc ((void **)&barg, N*sizeof(barg[0]));
 hipMalloc((void **)&xRes , D*sizeof(xRes[0]));  
// Copy A, b to the GPU.
 //hipMemcpy (cosArg, phase, NN*sizeof(phase[0]), hipMemcpyHostToDevice);
 //hipMemcpy (aaa, amp, NN*sizeof(aaa[0]), hipMemcpyHostToDevice);
 hipMemcpy (Aarg, A, N*D*sizeof(Aarg[0]), hipMemcpyHostToDevice);
 hipMemcpy (barg, b, N*sizeof(barg[0]), hipMemcpyHostToDevice);
 hipMemcpy (xRes, x, D*sizeof(xRes[0]), hipMemcpyHostToDevice);
 funcParams.res = xRes;
 funcParams.A = Aarg;
 funcParams.b = barg;
 funcParams.N = N;
 funcParams.D = D;
 
 hog_main<<<1,HOG_THREAD_CNT>>>(funcParams);
// "A" should now be in the array pointer "cosRes" on the device.
// We'll need to copy it to A
// "aaa", "cosArg" are NY by NX, while "cosRes" is NY by 2*NX
// (although everything here is stored in linear memory)
// Copy the result, which is A, from the device to the host
 hipMemcpy (x, xRes, D*sizeof(x[0]), hipMemcpyDeviceToHost);
// Done! */
// Free up the allocations on the CPU
 free(hostData);
 free(ArowCSR);
 
// Free up the allocations on the GPU
 hipFree(devData);
 hipFree(state);
 hipFree(gpuAval);
 hipFree(gpuArow);
 hipFree(gpuAcol);
 hipFree(gpuArowCSR);
 hipFree(gpux);
 hipFree(gpub);
}
